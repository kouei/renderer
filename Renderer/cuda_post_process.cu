#include "hip/hip_runtime.h"
#include<cstdint>
#include"cuda_post_process.h"

using namespace CUDA_POST_PROCESS;

__global__
void bgra_to_greyscale(const uchar4 * bgraImage, uchar4 * greyImage, int numRows, int numCols)
{
	//TODO
	//Fill in the kernel to convert from color to greyscale
	//the mapping from components of a uchar4 to RGBA is:
	// .x -> R ; .y -> G ; .z -> B ; .w -> A
	//
	//The output (greyImage) at each pixel should be the result of
	//applying the formula: output = .299f * R + .587f * G + .114f * B;
	//Note: We will be ignoring the alpha channel for this conversion

	//First create a mapping from the 2D block and grid locations
	//to an absolute 2D location in the image, then use that to
	//calculate a 1D offset
	auto r = blockIdx.y * blockDim.y + threadIdx.y;
	auto c = blockIdx.x * blockDim.x + threadIdx.x;
	if (r >= numRows || c >= numCols) return;

	auto pos_1D = r * numCols + c;
	auto color = bgraImage[pos_1D];
	auto intensity = static_cast<uint8_t>(0.299f * color.z + 0.587f * color.y + 0.114f * color.x);
	greyImage[pos_1D] = uchar4{ intensity, intensity, intensity, 0xff };
}

void CUDA_POST_PROCESS::your_bgra_to_greyscale(const void * h_in_bgraImage, void * h_out_bgraImage, int numRows, int numCols)
{
	auto mem_size = sizeof(uchar4) * numRows * numCols;
	void * d_bgraImage = nullptr;
	void * d_greyImage = nullptr;

	hipMalloc(&d_bgraImage, mem_size);
	hipMalloc(&d_greyImage, mem_size);

	hipMemcpy(d_bgraImage, h_in_bgraImage, mem_size, hipMemcpyHostToDevice);
	bgra_to_greyscale <<< dim3(128, 128), dim3(16, 16) >>> (reinterpret_cast<uchar4 *>(d_bgraImage), reinterpret_cast<uchar4 *>(d_greyImage), numRows, numCols);
	hipMemcpy(h_out_bgraImage, d_greyImage, mem_size, hipMemcpyDeviceToHost);

	hipFree(d_bgraImage);
	hipFree(d_greyImage);

	hipDeviceSynchronize(); 
	//checkCudaErrors(hipGetLastError());
}


__global__
void gaussian_blur(const uint8_t * inputChannel, uint8_t * outputChannel, int numRows, int numCols, const float * filter, int filterWidth)
{
	auto r = static_cast<int>(blockIdx.y * blockDim.y + threadIdx.y);
	auto c = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
	if (r >= numRows || c >= numCols) return;

	auto halfWidth = filterWidth / 2;

	auto sum = 0.0f;
	for (auto j = -halfWidth; j <= halfWidth; ++j)
	{
		for (auto i = -halfWidth; i <= halfWidth; ++i)
		{
			auto rr = r + j;
			auto cc = c + i;
			rr = rr > (numRows - 1) ? (numRows - 1) : (rr < 0 ? 0 : rr);
			cc = cc >(numCols - 1) ? (numCols - 1) : (cc < 0 ? 0 : cc);

			auto jj = j + halfWidth;
			auto ii = i + halfWidth;

			sum += static_cast<float>(inputChannel[rr * numCols + cc]) * filter[jj * filterWidth + ii];
		}
	}
	sum = sum > 255.0f ? 255.0f : (sum < 0.0f ? 0.0f : sum);
	outputChannel[r * numCols + c] = static_cast<uint8_t>(sum);
}
__global__
void separateChannels(const uchar4 * inputImageBGRA, int numRows, int numCols, uint8_t * redChannel, uint8_t * greenChannel, uint8_t * blueChannel)
{
	auto r = blockIdx.y * blockDim.y + threadIdx.y;
	auto c = blockIdx.x * blockDim.x + threadIdx.x;
	if (r >= numRows || c >= numCols) return;

	auto pos_1D = r * numCols + c;
	auto BGRA = inputImageBGRA[pos_1D];
	redChannel[pos_1D] = BGRA.z;
	greenChannel[pos_1D] = BGRA.y;
	blueChannel[pos_1D] = BGRA.x;
}


__global__
void recombineChannels(const uint8_t * redChannel, const uint8_t * greenChannel, const uint8_t * blueChannel, uchar4 * outputImageBGRA, int numRows, int numCols)
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	//make sure we don't try and access memory outside the image
	//by having any threads mapped there return early
	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows) return;

	auto red = redChannel[thread_1D_pos];
	auto green = greenChannel[thread_1D_pos];
	auto blue = blueChannel[thread_1D_pos];

	outputImageBGRA[thread_1D_pos] = make_uchar4(blue, green, red, 0xff);
}


void CUDA_POST_PROCESS::your_gaussian_blur(const void * h_inputImageBGRA, void * h_outputImageBGRA, int numRows, int numCols, float * h_filter, int filterWidth)
{
	void * d_inputImageBGRA = nullptr;
	void * d_outputImageBGRA = nullptr;
	void * d_red = nullptr;
	void * d_green = nullptr;
	void * d_blue = nullptr;
	void * d_redBlurred = nullptr;
	void * d_greenBlurred = nullptr;
	void * d_blueBlurred = nullptr;
	void * d_filter = nullptr;

	auto mem_size = sizeof(uchar4) * numRows * numCols;
	auto channel_mem_size = sizeof(uint8_t) * numRows * numCols;
	auto filter_mem_size = sizeof(float) * filterWidth * filterWidth;
	hipMalloc(&d_inputImageBGRA, mem_size);
	hipMalloc(&d_outputImageBGRA, mem_size);
	hipMalloc(&d_red, channel_mem_size);
	hipMalloc(&d_green, channel_mem_size);
	hipMalloc(&d_blue, channel_mem_size);
	hipMalloc(&d_redBlurred, channel_mem_size);
	hipMalloc(&d_greenBlurred, channel_mem_size);
	hipMalloc(&d_blueBlurred, channel_mem_size);
	hipMalloc(&d_filter, filter_mem_size);

	hipMemcpy(d_inputImageBGRA, h_inputImageBGRA, mem_size, hipMemcpyHostToDevice);
	hipMemcpy(d_filter, h_filter, filter_mem_size, hipMemcpyHostToDevice);

	separateChannels <<< dim3(128, 128), dim3(16, 16) >>> (reinterpret_cast<uchar4 *>(d_inputImageBGRA), numRows, numCols, reinterpret_cast<uint8_t *>(d_red), reinterpret_cast<uint8_t *>(d_green), reinterpret_cast<uint8_t *>(d_blue));

	hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());

	gaussian_blur <<< dim3(128, 128), dim3(16, 16) >>> (reinterpret_cast<uint8_t *>(d_red), reinterpret_cast<uint8_t *>(d_redBlurred), numRows, numCols, reinterpret_cast<float *>(d_filter), filterWidth);
	gaussian_blur <<< dim3(128, 128), dim3(16, 16) >>> (reinterpret_cast<uint8_t *>(d_green), reinterpret_cast<uint8_t *>(d_greenBlurred), numRows, numCols, reinterpret_cast<float *>(d_filter), filterWidth);
	gaussian_blur <<< dim3(128, 128), dim3(16, 16) >>> (reinterpret_cast<uint8_t *>(d_blue), reinterpret_cast<uint8_t *>(d_blueBlurred), numRows, numCols, reinterpret_cast<float *>(d_filter), filterWidth);



	hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());

	recombineChannels <<< dim3(128, 128), dim3(16, 16) >>> (reinterpret_cast<uint8_t *>(d_redBlurred), reinterpret_cast<uint8_t *>(d_greenBlurred), reinterpret_cast<uint8_t *>(d_blueBlurred), reinterpret_cast<uchar4 *>(d_outputImageBGRA), numRows, numCols);

	hipMemcpy(h_outputImageBGRA, d_outputImageBGRA, mem_size, hipMemcpyDeviceToHost);


	hipFree(d_inputImageBGRA);
	hipFree(d_outputImageBGRA);
	hipFree(d_red);
	hipFree(d_green);
	hipFree(d_blue);
	hipFree(d_redBlurred);
	hipFree(d_greenBlurred);
	hipFree(d_blueBlurred);
	hipFree(d_filter);

	hipDeviceSynchronize(); //checkCudaErrors(hipGetLastError());
}



__global__
void average_pooling(const uchar4 * d_input_buffer, uchar4 * d_output_buffer, int numRows, int numCols, int pooling_width)
{
	auto r = blockIdx.y * blockDim.y + threadIdx.y;
	auto c = blockIdx.x * blockDim.x + threadIdx.x;
	if (r >= numRows || c >= numCols) return;

	auto rr = r * pooling_width;
	auto cc = c * pooling_width;

	float avg_red = 0.0f;
	float avg_green = 0.0f;
	float avg_blue = 0.0f;

	for (auto j = 0; j < pooling_width; ++j)
	{
		for (auto i = 0; i < pooling_width; ++i)
		{
			auto BGRA = d_input_buffer[(rr + j) * numCols * pooling_width + cc + i];
			avg_red += BGRA.z;
			avg_green += BGRA.y;
			avg_blue += BGRA.x;
		}
	}
	auto total = pooling_width * pooling_width;
	avg_red /= total;
	avg_green /= total;
	avg_blue /= total;

	auto red = static_cast<uint8_t>(avg_red);
	auto green = static_cast<uint8_t>(avg_green);
	auto blue = static_cast<uint8_t>(avg_blue);

	d_output_buffer[r * numCols + c] = make_uchar4(blue, green, red, 0xff);
}

void CUDA_POST_PROCESS::your_average_pooling(const void * h_input_buffer, void * h_output_buffer, int numRows, int numCols, int pooling_width)
{
	void * d_input_buffer = nullptr;
	void * d_output_buffer = nullptr;
	auto output_mem_size = sizeof(uchar4) * numRows * numCols;
	auto input_mem_size = output_mem_size * pooling_width * pooling_width;

	hipMalloc(&d_input_buffer, input_mem_size);
	hipMalloc(&d_output_buffer, output_mem_size);

	hipMemcpy(d_input_buffer, h_input_buffer, input_mem_size, hipMemcpyHostToDevice);
	average_pooling <<< dim3(128, 128), dim3(16, 16) >>> (reinterpret_cast<uchar4 *>(d_input_buffer), reinterpret_cast<uchar4 *>(d_output_buffer), numRows, numCols, pooling_width);
	hipMemcpy(h_output_buffer, d_output_buffer, output_mem_size, hipMemcpyDeviceToHost);

	hipFree(d_input_buffer);
	hipFree(d_output_buffer);
}
